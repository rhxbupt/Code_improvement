#include "hip/hip_runtime.h"
/**
 * Naive Example of Matrix Addition
 *
 */

/**
 * Matrix multiplication: C = A + B.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

/**
 * Matrix addition (CUDA Kernel) on the device: C = A + B
 * w is matrix width, h is matrix height
 */
__global__ void
matrixAddCUDA(float *C, float *A, float *B, int w, int h)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread local index
    int txl = threadIdx.x;
    int tyl = threadIdx.y;

    // Thread global index
    int tx = txl+bx*blockDim.x;
    int ty = tyl+by*blockDim.y;
    int glbIdx = ty*w+tx;

    int maxidx = w*h-1;
    if (glbIdx<0 || glbIdx>maxidx){
      printf("Error: glbIdx is %d.\n", glbIdx);
    }
    else{
      // Do addition
      C[glbIdx] = A[glbIdx] + B[glbIdx];
    }
    // if (threadIdx.x==0 && threadIdx.y==0){
    //   printf("bx=%d, by=%d, txl=%d, tyl=%d, glbIdx=%d, A[glbIdx]=%f, B[glbIdx]=%f, C[glbIdx]=%f\n",
    // 	     bx, by, txl, tyl, glbIdx, A[glbIdx], B[glbIdx], C[glbIdx]);
    // }
}

void constantInit(float *data, int size)
{
    for (int i = 0; i < size; ++i)
    {
      data[i] = (float)rand()/RAND_MAX;
    }
}

int matrixAdd_gold(float *A, float *B, float*C, int size){
  for (int i=0;i<size;i++)
    C[i] = A[i] + B[i];
  return 0;
}

/**
 * A wrapper that calls the GPU kernel
 */
int matrixAdd(int block_size, int w, int h)
{
    // Allocate host memory for matrices A and B
  unsigned int sz = w*h;
  unsigned int mem_size = sizeof(float) * sz;
  float *h_A = (float *)malloc(mem_size);
  float *h_B = (float *)malloc(mem_size);
  float *h_C = (float *) malloc(mem_size);
  
    // Initialize host memory
    constantInit(h_A, sz);
    constantInit(h_B, sz);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipError_t error;
    error = hipMalloc((void **) &d_A, mem_size);
    error = hipMalloc((void **) &d_B, mem_size);
    error = hipMalloc((void **) &d_C, mem_size);
    
    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(w / threads.x, h / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    matrixAddCUDA<<< grid, threads >>>(d_C, d_A, d_B, w, h);

    printf("done\n");

    hipDeviceSynchronize();

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    /* check the result correctness */
    float g_sum=0, c_sum=0;
    for (int i=0;i<w*h;i++)      {
      //      if (fmod(i,32*w)==0) printf("h_C[%d]=%f\n",i,h_C[i]);
      g_sum += h_C[i];
    }
    matrixAdd_gold(h_A, h_B, h_C, w*h);
    for (int i=0;i<w*h;i++)       c_sum += h_C[i];    
    if (abs(g_sum - c_sum)<1e-10){
      printf("Pass...\n");
    }
    else{
      printf("Fail: %f vs. %f.\n", g_sum, c_sum);
    }
    
    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Matrix Addition Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
      //        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -w=Width -h=Height (Width x Height of Matrix)\n");
        printf("  Note: w and h should be multiples of 32, and neither shall exceed 1024.\n");

        exit(EXIT_SUCCESS);
    }

    int block_size = 32;

    int w=1024;
    int h=1024;

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "w"))
    {
        w = getCmdLineArgumentInt(argc, (const char **)argv, "w");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "h"))
    {
        h = getCmdLineArgumentInt(argc, (const char **)argv, "h");
    }

    if (w>1024 || h>1024 || fmod(w,32) || fmod(h,32))
    {
      printf("Error: w and h should be multiples of 32, and neither shall exceed 1024.\n");
      exit(EXIT_FAILURE);
    }

    printf("block_size=%d, matrix width=%d, matrix height=%d\n", block_size, w,h);

    int matrix_result = matrixAdd(block_size, w, h);

    exit(matrix_result);
}
